#include "hip/hip_runtime.h"

//header files
#include "hysortod.h"
#include "params.h"


//global constants


//prototypes
DTYPE getMax(DTYPE **dataset, int N);

DTYPE getMin(DTYPE **dataset, int N);


int main(int argc, char **argv)
{
	//initialize variables/datasets
	int i = 0, N, bins = 5;
	sscanf(argv[1],"%d",&N);	//number of lines in text
	DTYPE **dataset;
	char inputFname[500];
	strcpy(inputFname,argv[2]);	// the file name
	int cube = 0;

	//allocate memory for dataset
	dataset=(DTYPE**)malloc(sizeof(DTYPE*)*N);
	for (int i=0; i<N; i++)
	{
	dataset[i]=(DTYPE*)malloc(sizeof(DTYPE)*DIM);
	}

	//import the data set
	importDataset(inputFname, N, dataset);

	normalize(dataset, getMax(dataset, N), getMin(dataset,N), N);

	int arrSize = N;
	DTYPE *outlierArray = (DTYPE*)malloc(sizeof(DTYPE*)*arrSize);

	Hypercube **array = (Hypercube**)malloc(sizeof(Hypercube)*N);

	//initialize array to null initially
 	for(i = 0; i < N; i++)
	{
	array[i] = NULL;
	}
		

	//start the timer
	clock_t start, end;
	DTYPE cpu_time_used;
	start = clock();
	
	//perform outlier algorithm
	cube = HYsortOD(outlierArray, dataset, array, N, bins);

	//end timer
	end = clock();
	cpu_time_used = ((DTYPE) (end - start)) / CLOCKS_PER_SEC;
	
	//iterate through the outlier array
	for(int i = 0; i < cube; i++)
	{
			//print out the outlier score to that hypercube
			printf("%lf\n",outlierArray[i]);
			
	}
	
	  //free dataset
	  for (int i=0; i<N; i++)
	  {
		free(dataset[i]);
	  }
	  
	  free(dataset);  
	  free(outlierArray);	
	  
	printf("Time used : %f\n", cpu_time_used);		
	printf("End program!\n"); 
	return 0;
}

//implementation of prototypes
DTYPE getMax(DTYPE **dataset, int N)
{
	//initialize variables
	DTYPE max = 0.0;
	
	//processing
		//assume first one is max
		max = dataset[0][0];
		
		//loop through dataset to find max
		for(int i = 0; i < N; i++)
		{
			for(int j = 0; j < DIM; j++)
			{
				//check if the value is max value
				if(dataset[i][j] > max)
				{
					max = dataset[i][j];
				}
			}
		}
	
	//return the max
	return max;
}

DTYPE getMin(DTYPE **dataset, int N)
{
	//initialize variables
	DTYPE min = 0.0;
	
	//processing
		//assume first one is min
		min = dataset[0][0];
		
		//loop through dataset to find min
		for(int i = 0; i < N; i++)
		{
			for(int j = 0; j < DIM; j++)
			{
				//check if the value is min value
				if(dataset[i][j] > 0 && dataset[i][j] < min)
				{
					min = dataset[i][j];
				}
			}
		}
	
	//return the max
	return min;	
}
